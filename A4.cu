#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include<bits/stdc++.h> 
#include<cassert>
using namespace std;

struct tra{
    int numclasses;
    int src,dest;
    int* clas;
};

struct batch{
   int R;
   int* rid;
   int* tno;
   int* cls;
   int* src;
   int* dest;
   int* nseats;
   int* tid;
};

__global__ void parallel(int* darr,int* dstatus,batch request)//numof people,trainno,classno
{
    
  //for each request in a batch 
   int R=request.R;
   for(int i=0;i<R;i++)
   {
     
       if(request.tid[i]==blockIdx.x && request.cls[i]==threadIdx.x)
    {       //  printf("hey%d,%d\n",request.tno[i],request.tid[i]);
            int src=request.src[i];
            int dest=request.dest[i];
            
            int mini=darr[request.tno[i]*25*51+threadIdx.x*51+src];
            for(int k=src;k<dest;k++)
           {
              mini=min(mini,darr[request.tno[i]*25*51+threadIdx.x*51+k]);  
           }
          
          if(mini>=request.nseats[i])
          {
              dstatus[i]=1;
              for(int k=src;k<dest;k++)
              {
                  darr[request.tno[i]*25*51+threadIdx.x*51+k]-=request.nseats[i];
              }
          }
          else
          {
             dstatus[i]=0;
          }  
     }    
       
   }
   
}

void fn(int N,batch* b,int B,tra* trains)
{
    int* arr=(int*)malloc((N*25*51)*sizeof(int));//shared??
    int* darr;
    hipMalloc(&darr,(N*25*51)*sizeof(int));//array to see number of seats are vacent
    
    for(int i=0;i<N;i++)
    {
        int c=trains[i].numclasses;
        int p=trains[i].dest-trains[i].src;
        for(int j=0;j<c;j++)//jth class j<=25
        {
            int s=trains[i].clas[j];
            for(int k=0;k<=p;k++)
            {
                arr[i*25*51+j*51+k]=s;
            }
        }
    }
    
    hipMemcpy(darr,arr,(N*25*51)*sizeof(int),hipMemcpyHostToDevice);
    
    for(int i=0;i<B;i++)
    {
       map<int,int>mp;
       int tickets=0,s=0,f=0,mc=25;
       int R=b[i].R; 
       int *status,*dstatus;
       status=(int* )malloc(R*sizeof(int));
        hipMalloc(&dstatus,(R)*sizeof(int));

        batch gpb;
        hipMalloc(&gpb.rid,(R)*sizeof(int));
        hipMalloc(&gpb.tno,(R)*sizeof(int));
        hipMalloc(&gpb.tid,(R)*sizeof(int));
        hipMalloc(&gpb.cls,(R)*sizeof(int));
        hipMalloc(&gpb.src,(R)*sizeof(int));
        hipMalloc(&gpb.dest,(R)*sizeof(int));
        hipMalloc(&gpb.nseats,(R)*sizeof(int));
        
        
        hipMemcpy(gpb.rid,b[i].rid,R*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(gpb.cls,b[i].cls,R*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(gpb.src,b[i].src,R*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(gpb.dest,b[i].dest,R*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(gpb.nseats,b[i].nseats,R*sizeof(int),hipMemcpyHostToDevice);
        gpb.R=R;
        int x=1;
        for(int j=0;j<R;j++)
        {
          if(mp[b[i].tno[j]]==0)
          {
            mp[b[i].tno[j]]=x;
            x++;
          }
        }
        
        for(int j=0;j<R;j++)
        {
           b[i].tid[j]=mp[b[i].tno[j]];
        }
        
        hipMemcpy(gpb.tno,b[i].tno,R*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(gpb.tid,b[i].tid,R*sizeof(int),hipMemcpyHostToDevice);
        
        //parallel<<<N,mc>>>(darr,dstatus,gpb);
        parallel<<<R+1,mc>>>(darr,dstatus,gpb);
        hipDeviceSynchronize();
       hipMemcpy(status,dstatus,R*sizeof(int),hipMemcpyDeviceToHost);
        for(int j=0;j<R;j++)
        {
            if(status[j]==1)
            {
                printf("success\n");s++;
                tickets+=(b[i].dest[j]-b[i].src[j])*(b[i].nseats[j]);
            }
            else
            {
                printf("failure\n");f++;
            }
        }
        printf("%d %d\n",s,f);
        printf("%d\n",tickets);
    }
}


int main(int argc, char **argv)
{
  int N;
  scanf("%d", &N); // scaning for number of trains
  tra* trains=(tra *)malloc(N*sizeof(tra));//N trains                       //1
  for(int i=0;i<N;i++)
  {
      int tno,M,source,desti;
      scanf("%d", &tno);
      scanf("%d", &M);//no of classes
      scanf("%d", &source);//src
      scanf("%d", &desti);//dest
      if(source>desti)
      {
          int p=source;
          source=desti;
          desti=p;
      }
      trains[tno].src=source;
      trains[tno].dest=desti;
      trains[tno].numclasses=M;
      trains[tno].clas=(int*)malloc(M*sizeof(int));//m classes and this is to know max capacity in the given class
      for(int j=0;j<M;j++)
      {
          int cno,cap;
          scanf("%d", &cno);
          scanf("%d", &cap);
          trains[tno].clas[cno]=cap;
      }
  }
  
   int B;
   scanf("%d", &B);
   batch* b=(batch*)malloc(B*sizeof(batch));

   for(int i=0;i<B;i++)
   {
       int R;
       scanf("%d", &R);
       
       b[i].R=R;
       b[i].rid=(int*)malloc((R)*sizeof(int));
       b[i].tno=(int*)malloc((R)*sizeof(int));
       b[i].tid=(int*)malloc((R)*sizeof(int));
       b[i].cls=(int*)malloc((R)*sizeof(int));
       b[i].src=(int*)malloc((R)*sizeof(int));
       b[i].dest=(int*)malloc((R)*sizeof(int));
       b[i].nseats=(int*)malloc((R)*sizeof(int));
       
       for(int j=0;j<R;j++)
       {
           int rid,tno,cl,src,dest,n;
           scanf("%d", &rid);
           scanf("%d", &tno);
           scanf("%d", &cl);
           scanf("%d", &src);
           scanf("%d", &dest);
           scanf("%d", &n);
            if(src>dest){
               int p=src;
               src=dest;
               dest=p;
            }  
           b[i].rid[j]=rid;
           b[i].tno[j]=tno;
           b[i].cls[j]=cl;
           b[i].src[j]=src-trains[tno].src;
           b[i].dest[j]=dest-trains[tno].src;
           b[i].nseats[j]=n;
       }
   }
   fn(N,b,B,trains);
}


